#include "hip/hip_runtime.h"
#include <hip/device_functions.h>

#include "ma.h"

// Parameters for CUDA kernel executions
#define BLOCKX		16
#define BLOCKY		16
#define BLOCKSIZE	64
#define TILE_DIM	32
#define BLOCK_ROWS	8


/****** Global Variables *******/
short2 **maTextures;       // Two textures used to compute 2D Voronoi Diagram
int *dindex,*dreindex;
int *hindex,*hreindex;
int *pnt,*npnt,*pppl,*icpl;
int maMemSize;             // Size (in bytes) of a texture
int maTexSize;             // Texture size (squared texture)

texture<short2> maTexColor; 
texture<short2> maTexLinks;
texture<short2> maTexP1;
texture<short2> maTexIcp;
texture<short2> maTexPpp;
texture<int> maTexIndex;
texture<int> maTexReindex;
texture<int> maTexPnt;
texture<int> maTexNpnt;
texture<int> maTexPppl;
texture<int> maTexIcpl;

/********* Kernels ********/
#include "maKernel.h"

///////////////////////////////////////////////////////////////////////////
//
// Initialize necessary memory for 2D Voronoi Diagram computation
// - textureSize: The size of the Discrete Voronoi Diagram (width = height)
//
///////////////////////////////////////////////////////////////////////////

void makePattern(int step)
{
	int mstep=step,mstep2;
	int num=0;
	for (int i=0;i<maTexSize;i+=step) 
	{
		hindex[i]=num;
		hreindex[num]=i;
		num++;
	}
	while(mstep>=2)
	{
		mstep2 = mstep/2;
		for (int i=mstep2;i<maTexSize;i+=mstep)
		{
			hindex[i]=num;
			hreindex[num]=i;
			num++;
		}
		mstep = mstep2;
	}
	hipMemcpy(dindex,hindex,maTexSize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dreindex,hreindex,maTexSize*sizeof(int),hipMemcpyHostToDevice);
}

void maInitialization(int textureSize)
{
	maTexSize = textureSize; 
	maMemSize = maTexSize *( maTexSize ) * sizeof(short2) ; 

	maTextures = (short2 **) malloc(4 * sizeof(short2 *)); 

	// Allocate 2 textures
	hipMalloc((void **) &maTextures[0], maMemSize); 
	hipMalloc((void **) &maTextures[1], maMemSize); 
	hipMalloc((void **) &maTextures[2], maMemSize); 
	hipMalloc((void **) &maTextures[3], maMemSize); 
	hipMalloc((void **) &dindex, textureSize*sizeof(int)); 
	hipMalloc((void **) &dreindex, textureSize*sizeof(int));
	hipMalloc((void **) &pnt, textureSize*sizeof(int)); 
	hipMalloc((void **) &npnt, textureSize*sizeof(int)); 
	hipMalloc((void **) &pppl, textureSize*sizeof(int)); 
	hipMalloc((void **) &icpl, textureSize*sizeof(int)); 
	hindex = (int *) malloc(maTexSize*sizeof(int));
	hreindex = (int *) malloc(maTexSize*sizeof(int));
	hipMemset(icpl,-1,textureSize*sizeof(int));
	hipMemset(maTextures[2],127,maMemSize);
}

///////////////////////////////////////////////////////////////////////////
//
// Deallocate all allocated memory
//
///////////////////////////////////////////////////////////////////////////
void maDeInitialization()
{
	hipFree(maTextures[0]); 
	hipFree(maTextures[1]); 
	hipFree(maTextures[2]); 
	hipFree(maTextures[3]); 
	free(maTextures); 
	hipFree(dindex);
	hipFree(dreindex);
	hipFree(pnt);
	hipFree(npnt);
	hipFree(pppl);
	hipFree(icpl);
}


// Copy input to GPU 
void maInitializeInput(short *input)
{
	hipMemcpy(maTextures[0], input, maMemSize, hipMemcpyHostToDevice); 
}

// In-place transpose a squared texture. 
// Block orders are modified to optimize memory access. 
// Point coordinates are also swapped. 
/*void maTranspose(short2 *texture)
{
	dim3 block(TILE_DIM, BLOCK_ROWS); 
	dim3 grid(maTexSize / TILE_DIM, maTexSize / TILE_DIM); 

	hipBindTexture(0, maTexColor, texture); 
	maKernelTranspose<<< grid, block >>>(texture, maTexSize); 
	hipUnbindTexture(maTexColor); 
}*/
void maTranspose(short2 *texture)
{
	short *tmp1 = (short *)malloc(maMemSize);
	short *tmp2 = (short *)malloc(maMemSize);
	hipMemcpy(tmp1,texture,maMemSize,hipMemcpyDeviceToHost);
	for (int i=0;i<maTexSize;i++)
	{
		for (int j=0;j<maTexSize;j++)
		{
			int idx1 = i + j*maTexSize;
			int idx2 = j + i*maTexSize;
			tmp2[idx2*2] = tmp1[idx1*2+1];
			tmp2[idx2*2+1] = tmp1[idx1*2];
		}
	}
	hipMemcpy(texture,tmp2,maMemSize,hipMemcpyHostToDevice);
}

// Phase 1 of PBA. m1 must divides texture size
void maPhase1() 
{
	dim3 block = dim3(BLOCKSIZE);   
	dim3 grid = dim3((maTexSize / block.x)); 

	// Flood vertically in their own bands
	hipBindTexture(0, maTexColor, maTextures[0]); 
	maKernelFloodDown<<< grid, block >>>(maTextures[1], maTexSize); 
	hipUnbindTexture(maTexColor); 

	hipBindTexture(0, maTexColor, maTextures[1]); 
	maKernelFloodUp<<< grid, block >>>(maTextures[1], maTexSize); 
	hipUnbindTexture(maTexColor); 
}

// Phase 2 of PBA. m2 must divides texture size
void maPhase2(int STEP) 
{	
	int step = STEP;
	makePattern(step);
	hipBindTexture(0,maTexIndex,dindex);
	hipBindTexture(0,maTexReindex,dreindex);

	hipBindTexture(0,maTexColor,maTextures[1]);
	dim3 block,grid;
	block = dim3(BLOCKSIZE);
	grid = dim3(((maTexSize/STEP)/block.x) + 1 );
	maKernelColorInit1<<<grid,block>>>(maTextures[2],icpl,maTexSize,0,step);
	hipUnbindTexture(maTexColor);
	/*hipBindTexture(0,maTexIcp,maTextures[2]);
	hipBindTexture(0,maTexIcpl,icpl);
	maKernelColorInit2<<<grid,block>>>(maTextures[0],maTexSize,0,step);
	hipUnbindTexture(maTexIcpl);
	hipUnbindTexture(maTexIcp);*/

	while (step>=2)
	{
		int step2 = step/2;
		int taskNum = (maTexSize - step2)/step;
		taskNum++;

		block = dim3(BLOCKSIZE);
		grid = dim3((taskNum/BLOCKSIZE)  );
		hipBindTexture(0,maTexIcpl,icpl);
		hipBindTexture(0,maTexP1,maTextures[1]);
		hipBindTexture(0,maTexIcp,maTextures[2]);
		maKernelColor15<<<grid,block>>>(maTextures[3],pppl,maTexSize,step2,step);
		hipUnbindTexture(maTexIcp);
		hipUnbindTexture(maTexIcpl);
		hipUnbindTexture(maTexP1);


		//hipMemcpy(xxx, maTextures[5], maMemSize, hipMemcpyDeviceToHost); 

		block = dim3(BLOCKSIZE);
		grid = dim3((taskNum/BLOCKSIZE)   );
		hipBindTexture(0,maTexPppl,pppl);
		hipBindTexture(0,maTexPpp,maTextures[3]);
		maKernelColor2<<<grid,block>>>(maTextures[2],icpl,maTexSize,step2,step);
		hipUnbindTexture(maTexPppl);
		hipUnbindTexture(maTexPpp);




		block = dim3(BLOCKSIZE);
		grid = dim3((maTexSize/BLOCKSIZE)  );
		hipBindTexture(0,maTexIcp,maTextures[2]);
		hipBindTexture(0,maTexIcpl,icpl);
		maKernelColor3<<<grid,block>>>(maTextures[0],maTexSize);
		hipUnbindTexture(maTexIcp);
		hipUnbindTexture(maTexIcpl);
		step = step2;
	}

	hipUnbindTexture(maTexIndex);
	hipUnbindTexture(maTexReindex);

	/**
	int step = STEP;
	hipBindTexture(0,maTexColor,maTextures[1]);
	hipBindTexture(0,maTexIndex,index);
	dim3 block,grid;
	block = dim3(BLOCKSIZE);
	grid = dim3(((maTexSize/STEP)/block.x)+1);
	maKernelColorInit<<<grid,block>>>(maTextures[0],maTextures[2],maTexSize,0,step);

	while(step>=2)
	{
		int step2 = step/2;
		int taskNum = (maTexSize-step2)/step;
		taskNum+=1;
		block = dim3(BLOCKSIZE);
		grid = dim3((taskNum/BLOCKSIZE)+1);
		hipBindTexture(0,maTexLinks,maTextures[0]);
		maKernelColorLine<<<grid,block>>>(maTextures[0],maTextures[2],maTexSize,step2,step);
		hipUnbindTexture(maTexLinks);
		step=step2;
	}

	hipUnbindTexture(maTexColor);
	hipUnbindTexture(maTexIndex);
	/**/
	/**
	hipBindTexture(0,maTexColor,maTextures[1]);
	dim3 block,grid;
	block = dim3(BLOCKSIZE);
	grid = dim3(maTexSize/block.x);
	maKernelTest1<<<grid,block>>>(maTextures[2],index,maTexSize);
	hipBindTexture(0,maTexLinks,maTextures[2]);
	hipBindTexture(0,maTexIndex,index);
	maKernelTest2<<<grid,block>>>(maTextures[0],maTexSize);
	hipUnbindTexture(maTexColor);
	hipUnbindTexture(maTexLinks);
	hipUnbindTexture(maTexIndex);/**/
}

void maCompute(int STEP,short* output)
{
	//hipMemcpy(xx, maTextures[0], maMemSize, hipMemcpyDeviceToHost);
	maPhase1(); 
//hipMemcpy(xx, maTextures[1], maMemSize, hipMemcpyDeviceToHost); 
	maTranspose(maTextures[1]); 
//	hipMemcpy(xx, maTextures[1], maMemSize, hipMemcpyDeviceToHost); 
	maPhase2(STEP); 

	maTranspose(maTextures[0]); 
	
}

void rerange(short *outData)
{
	short *tmp = (short *)malloc(maMemSize);
	for (int i=0;i<maTexSize;i++)
	{
		int k = hreindex[i];
		for (int j=0;j<maTexSize;j++)
		{
			int idtmp = j + k*maTexSize;
			int id = j+ i*maTexSize;
			tmp[idtmp*2] = outData[id*2];
			tmp[idtmp*2+1] = outData[id*2+1];
		}
	}
	memcpy(outData,tmp,maMemSize);
}

// Compute 2D Voronoi diagram
// Input: a 2D texture. Each pixel is represented as two "short" integer. 
//    For each site at (x, y), the pixel at coordinate (x, y) should contain 
//    the pair (x, y). Pixels that are not sites should contain the pair (MARKER, MARKER)
// See original paper for the effect of the three parameters: 
//    phase1Band, phase2Band, phase3Band
// Parameters must divide textureSize
void ma(short *input, short *output, int STEP) 
{
	// Initialization
	maInitializeInput(input); 
	makePattern(STEP);
	// Computation
	maCompute(STEP,output); 

	// Copy back the result
	hipMemcpy(output, maTextures[0], maMemSize, hipMemcpyDeviceToHost); 
	//rerange(output);
	
}